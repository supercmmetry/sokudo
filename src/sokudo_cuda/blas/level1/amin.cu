#include "hip/hip_runtime.h"
#include <hipblas.h>
#include "amin.h"

CudaAbstractTask cu_samin(float *a, uint64_t *res, uint64_t n, uint64_t incx) {
    CudaError err;
    CublasError berr;
    hipblasHandle_t handle;
    berr << hipblasCreate(&handle);

    hipStream_t stream;
    err << hipStreamCreate(&stream);

    berr << hipblasSetStream(handle, stream);

    float *dx;
    err << hipMalloc(&dx, n * sizeof(float));
    err << hipMemcpyAsync(dx, a, n * sizeof(float), hipMemcpyHostToDevice, stream);
    auto p = n / incx + (n % incx != 0);
    int r = 0;
    berr << hipblasIsamin(handle, p, dx, incx, &r);
    *res = r;
    return CudaAbstractTask(stream, handle) << dx;
}

CudaAbstractTask cu_damin(double *a, uint64_t *res, uint64_t n, uint64_t incx) {
    CudaError err;
    CublasError berr;
    hipblasHandle_t handle;
    berr << hipblasCreate(&handle);

    hipStream_t stream;
    err << hipStreamCreate(&stream);

    berr << hipblasSetStream(handle, stream);

    double *dx;
    err << hipMalloc(&dx, n * sizeof(double));
    err << hipMemcpyAsync(dx, a, n * sizeof(double), hipMemcpyHostToDevice, stream);
    auto p = n / incx + (n % incx != 0);
    int r = 0;
    berr << hipblasIdamin(handle, p, dx, incx, &r);
    *res = r;
    return CudaAbstractTask(stream, handle) << dx;
}

CudaAbstractTask cu_scamin(void *a, uint64_t *res, uint64_t n, uint64_t incx) {
    CudaError err;
    CublasError berr;
    hipblasHandle_t handle;
    berr << hipblasCreate(&handle);

    hipStream_t stream;
    err << hipStreamCreate(&stream);

    berr << hipblasSetStream(handle, stream);

    double *dx;
    err << hipMalloc(&dx, n * sizeof(float2));
    err << hipMemcpyAsync(dx, a, n * sizeof(float2), hipMemcpyHostToDevice, stream);
    auto p = n / incx + (n % incx != 0);
    int r = 0;
    berr << hipblasIcamin(handle, p, reinterpret_cast<const hipComplex *>(dx), incx, &r);
    *res = r;
    return CudaAbstractTask(stream, handle) << dx;
}

CudaAbstractTask cu_dcamin(void *a, uint64_t *res, uint64_t n, uint64_t incx) {
    CudaError err;
    CublasError berr;
    hipblasHandle_t handle;
    berr << hipblasCreate(&handle);

    hipStream_t stream;
    err << hipStreamCreate(&stream);

    berr << hipblasSetStream(handle, stream);

    double *dx;
    err << hipMalloc(&dx, n * sizeof(double2));
    err << hipMemcpyAsync(dx, a, n * sizeof(double2), hipMemcpyHostToDevice, stream);
    auto p = n / incx + (n % incx != 0);
    int r = 0;
    berr << hipblasIzamin(handle, p, reinterpret_cast<const hipDoubleComplex *>(dx), incx, &r);
    *res = r;
    return CudaAbstractTask(stream, handle) << dx;
}
