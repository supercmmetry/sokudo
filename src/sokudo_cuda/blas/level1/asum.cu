#include "hip/hip_runtime.h"
#include <hipblas.h>
#include "asum.h"

CudaAbstractTask cu_sasum(float *a, float *res, uint64_t n, uint64_t incx, uint64_t size) {
    CudaError err;
    CublasError berr;
    hipblasHandle_t handle;
    berr << hipblasCreate(&handle);

    hipStream_t stream;
    err << hipStreamCreate(&stream);

    berr << hipblasSetStream(handle, stream);

    float *dx;
    err << hipMalloc(&dx, size * sizeof(float));
    err << hipMemcpyAsync(dx, a, size * sizeof(float), hipMemcpyHostToDevice, stream);
    berr << hipblasSasum(handle, n, dx, incx, res);
    return CudaAbstractTask(stream, handle) << dx;
}

CudaAbstractTask cu_dasum(double *a, double *res, uint64_t n, uint64_t incx, uint64_t size) {
    CudaError err;
    CublasError berr;
    hipblasHandle_t handle;
    berr << hipblasCreate(&handle);

    hipStream_t stream;
    err << hipStreamCreate(&stream);

    berr << hipblasSetStream(handle, stream);

    double *dx;
    err << hipMalloc(&dx, size * sizeof(double));
    err << hipMemcpyAsync(dx, a, size * sizeof(double), hipMemcpyHostToDevice, stream);
    berr << hipblasDasum(handle, n, dx, incx, res);
    return CudaAbstractTask(stream, handle) << dx;
}

CudaAbstractTask cu_scasum(void *a, float *res, uint64_t n, uint64_t incx, uint64_t size) {
    CudaError err;
    CublasError berr;
    hipblasHandle_t handle;
    berr << hipblasCreate(&handle);

    hipStream_t stream;
    err << hipStreamCreate(&stream);

    berr << hipblasSetStream(handle, stream);

    double *dx;
    err << hipMalloc(&dx, size * sizeof(float2));
    err << hipMemcpyAsync(dx, a, size * sizeof(float2), hipMemcpyHostToDevice, stream);
    berr << hipblasScasum(handle, n, reinterpret_cast<const hipComplex *>(dx), incx, res);
    return CudaAbstractTask(stream, handle) << dx;
}

CudaAbstractTask cu_dcasum(void *a, double *res, uint64_t n, uint64_t incx, uint64_t size) {
    CudaError err;
    CublasError berr;
    hipblasHandle_t handle;
    berr << hipblasCreate(&handle);

    hipStream_t stream;
    err << hipStreamCreate(&stream);

    berr << hipblasSetStream(handle, stream);

    double *dx;
    err << hipMalloc(&dx, size * sizeof(double2));
    err << hipMemcpyAsync(dx, a, size * sizeof(double2), hipMemcpyHostToDevice, stream);
    berr << hipblasDzasum(handle, n, reinterpret_cast<const hipDoubleComplex *>(dx), incx, res);
    return CudaAbstractTask(stream, handle) << dx;
}
