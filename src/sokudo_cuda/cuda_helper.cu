
#include "cuda_helper.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <>

CudaAbstractTask::CudaAbstractTask(void *stream, void *cublas_handle) {
    _stream = stream;
    _cublas_handle = cublas_handle;
}

void CudaAbstractTask::destroy() {
    CudaError err;
    CublasError berr;
    while(_top) {
        err << hipFree(_top->device_ptr);
        CudaAllocation *tmp = _top;
        _top = _top->prev;
        delete tmp;
    }

    err << hipStreamDestroy((hipStream_t) _stream);
    if (_cublas_handle) {
        berr << hipblasDestroy((hipblasHandle_t) _cublas_handle);
    }
}

void CudaAbstractTask::sync() {
    if (hipStreamSynchronize((hipStream_t) _stream) != hipSuccess) {
        throw CudaException("hipStreamSynchronize failed");
    }
}

void CudaError::operator<<(int err) {
    auto error = (hipError_t) err;
    if (error != hipSuccess) {
        throw CudaException(std::string("CUDA operation failed with error=") + std::to_string(err));
    }
}

void CublasError::operator<<(int err) {
    auto error = (hipblasStatus_t) err;
    if (error != HIPBLAS_STATUS_SUCCESS) {
        throw CudaException(std::string("cuBLAS operation failed with error=") + std::to_string(err));
    }
}
