#include <hipblas.h>
#include "asum.h"

CudaAbstractTask cu_sasum(float *a, float *res, uint64_t n) {
    CudaError err;
    CublasError berr;
    hipblasHandle_t handle;
    berr << hipblasCreate(&handle);

    hipStream_t stream;
    err << hipStreamCreate(&stream);

    berr << hipblasSetStream(handle, stream);

    float *dx;
    err << hipMalloc(&dx, n * sizeof(float));
    err << hipMemcpyAsync(dx, a, n * sizeof(float), hipMemcpyHostToDevice, stream);

    berr << hipblasSasum(handle, n, dx, 1, res);
    return CudaAbstractTask(stream, handle) << dx;
}
