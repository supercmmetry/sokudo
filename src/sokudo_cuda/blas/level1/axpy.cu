#include <hipblas.h>
#include "axpy.h"

#define MIN(a, b) ((a) < (b) ? (a) : (b))

CudaAbstractTask cu_saxpy(float *alpha, float *x, uint64_t incx, float *y, uint64_t incy, uint64_t n) {
    CudaError err;
    CublasError berr;
    hipblasHandle_t handle;
    berr << hipblasCreate(&handle);

    hipStream_t stream;
    err << hipStreamCreate(&stream);

    berr << hipblasSetStream(handle, stream);

    float *dx, *dy;
    err << hipMalloc(&dx, n * sizeof(float));
    err << hipMalloc(&dy, n * sizeof(float));
    err << hipMemcpyAsync(dx, x, n * sizeof(float), hipMemcpyHostToDevice, stream);
    err << hipMemcpyAsync(dy, y, n * sizeof(float), hipMemcpyHostToDevice, stream);
    auto p1 = n / incx + (n % incx != 0);
    auto p2 = n / incy + (n % incy != 0);
    berr << hipblasSaxpy(handle, MIN(p1, p2), alpha, dx, incx, dy, incy);
    err << hipMemcpyAsync(y, dy, n * sizeof(float), hipMemcpyDeviceToHost, stream);
    return CudaAbstractTask(stream, handle) << dx << dy;
}

CudaAbstractTask cu_daxpy(double *alpha, double *x, uint64_t incx, double *y, uint64_t incy, uint64_t n) {
    CudaError err;
    CublasError berr;
    hipblasHandle_t handle;
    berr << hipblasCreate(&handle);

    hipStream_t stream;
    err << hipStreamCreate(&stream);

    berr << hipblasSetStream(handle, stream);

    double *dx, *dy;
    err << hipMalloc(&dx, n * sizeof(double));
    err << hipMalloc(&dy, n * sizeof(double));
    err << hipMemcpyAsync(dx, x, n * sizeof(double), hipMemcpyHostToDevice, stream);
    err << hipMemcpyAsync(dy, y, n * sizeof(double), hipMemcpyHostToDevice, stream);
    auto p1 = n / incx + (n % incx != 0);
    auto p2 = n / incy + (n % incy != 0);
    berr << hipblasDaxpy(handle, MIN(p1, p2), alpha, dx, incx, dy, incy);
    err << hipMemcpyAsync(y, dy, n * sizeof(double), hipMemcpyDeviceToHost, stream);
    return CudaAbstractTask(stream, handle) << dx << dy;
}

CudaAbstractTask cu_scaxpy(void *alpha, void *x, uint64_t incx, void *y, uint64_t incy, uint64_t n) {
    CudaError err;
    CublasError berr;
    hipblasHandle_t handle;
    berr << hipblasCreate(&handle);

    hipStream_t stream;
    err << hipStreamCreate(&stream);

    berr << hipblasSetStream(handle, stream);

    hipComplex *dx, *dy;
    err << hipMalloc(&dx, n * sizeof(hipComplex));
    err << hipMalloc(&dy, n * sizeof(hipComplex));
    err << hipMemcpyAsync(dx, x, n * sizeof(hipComplex), hipMemcpyHostToDevice, stream);
    err << hipMemcpyAsync(dy, y, n * sizeof(hipComplex), hipMemcpyHostToDevice, stream);
    auto p1 = n / incx + (n % incx != 0);
    auto p2 = n / incy + (n % incy != 0);
    berr << hipblasCaxpy(handle, MIN(p1, p2), (hipComplex*) alpha, dx, incx, dy, incy);
    err << hipMemcpyAsync(y, dy, n * sizeof(hipComplex), hipMemcpyDeviceToHost, stream);
    return CudaAbstractTask(stream, handle) << dx << dy;
}

CudaAbstractTask cu_dcaxpy(void *alpha, void *x, uint64_t incx, void *y, uint64_t incy, uint64_t n) {
    CudaError err;
    CublasError berr;
    hipblasHandle_t handle;
    berr << hipblasCreate(&handle);

    hipStream_t stream;
    err << hipStreamCreate(&stream);

    berr << hipblasSetStream(handle, stream);

    hipDoubleComplex *dx, *dy;
    err << hipMalloc(&dx, n * sizeof(hipDoubleComplex));
    err << hipMalloc(&dy, n * sizeof(hipDoubleComplex));
    err << hipMemcpyAsync(dx, x, n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream);
    err << hipMemcpyAsync(dy, y, n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream);
    auto p1 = n / incx + (n % incx != 0);
    auto p2 = n / incy + (n % incy != 0);
    berr << hipblasZaxpy(handle, MIN(p1, p2), (hipDoubleComplex*) alpha, dx, incx, dy, incy);
    err << hipMemcpyAsync(y, dy, n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost, stream);
    return CudaAbstractTask(stream, handle) << dx << dy;
}
