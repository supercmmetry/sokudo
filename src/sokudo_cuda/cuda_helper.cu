
#include "cuda_helper.h"
#include <hip/hip_runtime.h>
#include <>

CudaAbstractTask::CudaAbstractTask(void *stream) {
    _stream = stream;
}

void CudaAbstractTask::destroy() {
    while(_top) {
        hipFree(_top->device_ptr);
        CudaAllocation *tmp = _top;
        _top = _top->prev;
        delete tmp;
    }

    hipStreamDestroy((hipStream_t) _stream);
}

void CudaAbstractTask::sync() {
    if (hipStreamSynchronize((hipStream_t) _stream) != hipSuccess) {
        throw CudaException("hipStreamSynchronize failed");
    }
}
