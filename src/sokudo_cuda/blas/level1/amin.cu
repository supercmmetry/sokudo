#include "hip/hip_runtime.h"
#include <hipblas.h>
#include "amin.h"

CudaAbstractTask cu_samin(float *a, uint64_t *res, uint64_t n, uint64_t incx, uint64_t size) {
    CudaError err;
    CublasError berr;
    hipblasHandle_t handle;
    berr << hipblasCreate(&handle);

    hipStream_t stream;
    err << hipStreamCreate(&stream);

    berr << hipblasSetStream(handle, stream);

    float *dx;
    err << hipMalloc(&dx, size * sizeof(float));
    err << hipMemcpyAsync(dx, a, size * sizeof(float), hipMemcpyHostToDevice, stream);
    int r = 0;
    berr << hipblasIsamin(handle, n, dx, incx, &r);
    *res = r;
    return CudaAbstractTask(stream, handle) << dx;
}

CudaAbstractTask cu_damin(double *a, uint64_t *res, uint64_t n, uint64_t incx, uint64_t size) {
    CudaError err;
    CublasError berr;
    hipblasHandle_t handle;
    berr << hipblasCreate(&handle);

    hipStream_t stream;
    err << hipStreamCreate(&stream);

    berr << hipblasSetStream(handle, stream);

    double *dx;
    err << hipMalloc(&dx, size * sizeof(double));
    err << hipMemcpyAsync(dx, a, size * sizeof(double), hipMemcpyHostToDevice, stream);
    int r = 0;
    berr << hipblasIdamin(handle, n, dx, incx, &r);
    *res = r;
    return CudaAbstractTask(stream, handle) << dx;
}

CudaAbstractTask cu_scamin(void *a, uint64_t *res, uint64_t n, uint64_t incx, uint64_t size) {
    CudaError err;
    CublasError berr;
    hipblasHandle_t handle;
    berr << hipblasCreate(&handle);

    hipStream_t stream;
    err << hipStreamCreate(&stream);

    berr << hipblasSetStream(handle, stream);

    double *dx;
    err << hipMalloc(&dx, size * sizeof(float2));
    err << hipMemcpyAsync(dx, a, size * sizeof(float2), hipMemcpyHostToDevice, stream);
    int r = 0;
    berr << hipblasIcamin(handle, n, reinterpret_cast<const hipComplex *>(dx), incx, &r);
    *res = r;
    return CudaAbstractTask(stream, handle) << dx;
}

CudaAbstractTask cu_dcamin(void *a, uint64_t *res, uint64_t n, uint64_t incx, uint64_t size) {
    CudaError err;
    CublasError berr;
    hipblasHandle_t handle;
    berr << hipblasCreate(&handle);

    hipStream_t stream;
    err << hipStreamCreate(&stream);

    berr << hipblasSetStream(handle, stream);

    double *dx;
    err << hipMalloc(&dx, size * sizeof(double2));
    err << hipMemcpyAsync(dx, a, size * sizeof(double2), hipMemcpyHostToDevice, stream);
    int r = 0;
    berr << hipblasIzamin(handle, n, reinterpret_cast<const hipDoubleComplex *>(dx), incx, &r);
    *res = r;
    return CudaAbstractTask(stream, handle) << dx;
}
