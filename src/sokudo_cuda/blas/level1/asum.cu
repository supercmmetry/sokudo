#include "hip/hip_runtime.h"
#include <hipblas.h>
#include "asum.h"

CudaAbstractTask cu_sasum(float *a, float *res, uint64_t n, uint64_t incx) {
    CudaError err;
    CublasError berr;
    hipblasHandle_t handle;
    berr << hipblasCreate(&handle);

    hipStream_t stream;
    err << hipStreamCreate(&stream);

    berr << hipblasSetStream(handle, stream);

    float *dx;
    err << hipMalloc(&dx, n * sizeof(float));
    err << hipMemcpyAsync(dx, a, n * sizeof(float), hipMemcpyHostToDevice, stream);
    auto p = n / incx + (n % incx != 0);
    berr << hipblasSasum(handle, p, dx, incx, res);
    return CudaAbstractTask(stream, handle) << dx;
}

CudaAbstractTask cu_dasum(double *a, double *res, uint64_t n, uint64_t incx) {
    CudaError err;
    CublasError berr;
    hipblasHandle_t handle;
    berr << hipblasCreate(&handle);

    hipStream_t stream;
    err << hipStreamCreate(&stream);

    berr << hipblasSetStream(handle, stream);

    double *dx;
    err << hipMalloc(&dx, n * sizeof(double));
    err << hipMemcpyAsync(dx, a, n * sizeof(double), hipMemcpyHostToDevice, stream);
    auto p = n / incx + (n % incx != 0);
    berr << hipblasDasum(handle, p, dx, incx, res);
    return CudaAbstractTask(stream, handle) << dx;
}

CudaAbstractTask cu_scasum(float2 *a, float *res, uint64_t n, uint64_t incx) {
    CudaError err;
    CublasError berr;
    hipblasHandle_t handle;
    berr << hipblasCreate(&handle);

    hipStream_t stream;
    err << hipStreamCreate(&stream);

    berr << hipblasSetStream(handle, stream);

    double *dx;
    err << hipMalloc(&dx, n * sizeof(double));
    err << hipMemcpyAsync(dx, a, n * sizeof(double), hipMemcpyHostToDevice, stream);
    auto p = n / incx + (n % incx != 0);
    berr << hipblasScasum(handle, p, reinterpret_cast<const hipComplex *>(dx), incx, res);
    return CudaAbstractTask(stream, handle) << dx;
}
