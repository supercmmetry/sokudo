#include "hip/hip_runtime.h"
#include "cuda_test.h"
#include <hip/hip_runtime.h>
#include <>

__global__ void add(int *a, int *b) {
    int index = threadIdx.x;
    b[index] += a[index];
}


void cu_add_test(int *a, int *b, int n) {
    int *da, *db;
    hipMalloc(&da, n * sizeof(int));
    hipMalloc(&db, n * sizeof(int));
    hipMemcpy(da, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(db, b, n * sizeof(int), hipMemcpyHostToDevice);

    add<<<1, n>>>(da, db);
    hipDeviceSynchronize();

    hipMemcpy(b, db, sizeof(int) * n, hipMemcpyDeviceToHost);
}
