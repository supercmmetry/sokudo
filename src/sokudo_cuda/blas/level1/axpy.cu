#include <hipblas.h>
#include "axpy.h"

CudaAbstractTask
cu_saxpy(float *alpha, float *x, uint64_t incx, float *y, uint64_t incy, uint64_t n, uint64_t size_x, uint64_t size_y) {
    CudaError err;
    CublasError berr;
    hipblasHandle_t handle;
    berr << hipblasCreate(&handle);

    hipStream_t stream;
    err << hipStreamCreate(&stream);

    berr << hipblasSetStream(handle, stream);

    float *dx, *dy;
    err << hipMalloc(&dx, size_x * sizeof(float));
    err << hipMalloc(&dy, size_y * sizeof(float));
    err << hipMemcpyAsync(dx, x, size_x * sizeof(float), hipMemcpyHostToDevice, stream);
    err << hipMemcpyAsync(dy, y, size_y * sizeof(float), hipMemcpyHostToDevice, stream);
    berr << hipblasSaxpy(handle, n, alpha, dx, incx, dy, incy);
    err << hipMemcpyAsync(y, dy, size_y * sizeof(float), hipMemcpyDeviceToHost, stream);
    return CudaAbstractTask(stream, handle) << dx << dy;
}

CudaAbstractTask
cu_daxpy(double *alpha, double *x, uint64_t incx, double *y, uint64_t incy, uint64_t n, uint64_t size_x,
         uint64_t size_y) {
    CudaError err;
    CublasError berr;
    hipblasHandle_t handle;
    berr << hipblasCreate(&handle);

    hipStream_t stream;
    err << hipStreamCreate(&stream);

    berr << hipblasSetStream(handle, stream);

    double *dx, *dy;
    err << hipMalloc(&dx, size_x * sizeof(double));
    err << hipMalloc(&dy, size_y * sizeof(double));
    err << hipMemcpyAsync(dx, x, size_x * sizeof(double), hipMemcpyHostToDevice, stream);
    err << hipMemcpyAsync(dy, y, size_y * sizeof(double), hipMemcpyHostToDevice, stream);
    berr << hipblasDaxpy(handle, n, alpha, dx, incx, dy, incy);
    err << hipMemcpyAsync(y, dy, size_y * sizeof(double), hipMemcpyDeviceToHost, stream);
    return CudaAbstractTask(stream, handle) << dx << dy;
}

CudaAbstractTask
cu_scaxpy(void *alpha, void *x, uint64_t incx, void *y, uint64_t incy, uint64_t n, uint64_t size_x, uint64_t size_y) {
    CudaError err;
    CublasError berr;
    hipblasHandle_t handle;
    berr << hipblasCreate(&handle);

    hipStream_t stream;
    err << hipStreamCreate(&stream);

    berr << hipblasSetStream(handle, stream);

    hipComplex *dx, *dy;
    err << hipMalloc(&dx, size_x * sizeof(hipComplex));
    err << hipMalloc(&dy, size_y * sizeof(hipComplex));
    err << hipMemcpyAsync(dx, x, size_x * sizeof(hipComplex), hipMemcpyHostToDevice, stream);
    err << hipMemcpyAsync(dy, y, size_y * sizeof(hipComplex), hipMemcpyHostToDevice, stream);
    berr << hipblasCaxpy(handle, n, (hipComplex *) alpha, dx, incx, dy, incy);
    err << hipMemcpyAsync(y, dy, size_y * sizeof(hipComplex), hipMemcpyDeviceToHost, stream);
    return CudaAbstractTask(stream, handle) << dx << dy;
}

CudaAbstractTask
cu_dcaxpy(void *alpha, void *x, uint64_t incx, void *y, uint64_t incy, uint64_t n, uint64_t size_x, uint64_t size_y) {
    CudaError err;
    CublasError berr;
    hipblasHandle_t handle;
    berr << hipblasCreate(&handle);

    hipStream_t stream;
    err << hipStreamCreate(&stream);

    berr << hipblasSetStream(handle, stream);

    hipDoubleComplex *dx, *dy;
    err << hipMalloc(&dx, size_x * sizeof(hipDoubleComplex));
    err << hipMalloc(&dy, size_y * sizeof(hipDoubleComplex));
    err << hipMemcpyAsync(dx, x, size_x * sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream);
    err << hipMemcpyAsync(dy, y, size_y * sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream);
    berr << hipblasZaxpy(handle, n, (hipDoubleComplex *) alpha, dx, incx, dy, incy);
    err << hipMemcpyAsync(y, dy, size_y * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost, stream);
    return CudaAbstractTask(stream, handle) << dx << dy;
}
