#include "hip/hip_runtime.h"
#include "cuda_test.h"
#include <hip/hip_runtime.h>
#include <>

__global__ void add(const int *a, int *b) {
    auto index = threadIdx.x;
    b[index] += a[index];
}


CudaAbstractTask cu_add_test(int *a, int *b, int n) {
    CudaError err;
    hipStream_t stream;
    hipStreamCreate(&stream);

    int *da, *db;
    err << hipMalloc(&da, n * sizeof(int));
    err << hipMalloc(&db, n * sizeof(int));
    err << hipMemcpyAsync(da, a, n * sizeof(int), hipMemcpyHostToDevice, stream);
    err << hipMemcpyAsync(db, b, n * sizeof(int), hipMemcpyHostToDevice, stream);

    add<<<1, n, 0, stream>>>(da, db);

    err << hipMemcpyAsync(b, db, sizeof(int) * n, hipMemcpyDeviceToHost, stream);

    return CudaAbstractTask(stream) << da << db;
}
