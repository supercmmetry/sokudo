#include "hip/hip_runtime.h"
#include "cuda_test.h"
#include <hip/hip_runtime.h>
#include <>

__global__ void add(const int *a, int *b) {
    auto index = threadIdx.x;
    b[index] += a[index];
}


CudaAbstractTask cu_add_test(int *a, int *b, int n) {
    hipStream_t stream;
    if (hipStreamCreate(&stream) != hipSuccess) {
        throw CudaException("hipStreamCreate failed");
    }

    int *da, *db;
    hipMalloc(&da, n * sizeof(int));
    hipMalloc(&db, n * sizeof(int));
    hipMemcpyAsync(da, a, n * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(db, b, n * sizeof(int), hipMemcpyHostToDevice, stream);

    add<<<1, n, 0, stream>>>(da, db);

    hipMemcpyAsync(b, db, sizeof(int) * n, hipMemcpyDeviceToHost, stream);

    return CudaAbstractTask(stream) << da << db;
}
